#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <hip/hip_runtime.h>
#include <cmath>
#include "../solve/tools.h"
#include "unitTest.h"
#include <cmath>
void testFillb(int nx, int ny, double dt, double dx, double dy, double *u, double *v, double *Yn, double *b_expeted, double *b)
{
    // check if dx and dy are  positive
    if (dx <= 0 || dy <= 0)
    {
        throw std::invalid_argument("dx and dy must be positive.");
    }
    // Malloc in gpu varaibles needed
    double *d_u, *d_v, *d_Yn;
    double *d_b;

    hipMalloc(&d_b, nx * ny * sizeof(double));
    hipMalloc(&d_u, nx * ny * sizeof(double));
    hipMalloc(&d_Yn, nx * ny * sizeof(double));
    hipMalloc(&d_v, nx * ny * sizeof(double));

    hipMemcpy(d_u, u, nx * ny * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_v, v, nx * ny * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Yn, Yn, nx * ny * sizeof(double), hipMemcpyHostToDevice);
    // init kernel
    dim3 blockDim(16, 18);
    dim3 gridDim((nx + blockDim.x - 1) / blockDim.x, (ny + blockDim.y - 1) / blockDim.y);
    computeB<<<gridDim, blockDim>>>(d_b, d_Yn, d_u, d_v, dx, dy, nx, ny, dt);
    hipMemcpy(b, d_b, nx * ny * sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    for (int i = 0; i < ny; i++)
    {
        for (int j = 0; j < nx; j++)
        {
            int idx = i * ny + j;
            assert(fabs(b_expeted[idx] - b[idx]) < 0.2); // for double values is necessary a tolerance
        }
    }
    //== Free CUDA memory
    hipFree(d_b);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_Yn);
}