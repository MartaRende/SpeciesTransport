#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <hip/hip_runtime.h>
#include <cmath>
#include "../solve/tools.h"
#include "unitTest.h"
#include <cmath>
void testFillb(int nx, int ny, double dt, double dx, double dy, double *u, double *v, double *Yn, double *b_expeted, double *b)
{

    double *d_u, *d_v, *d_Yn;
    double *d_b;

    hipMalloc(&d_b, nx * ny * sizeof(double));
    hipMalloc(&d_u, nx * ny * sizeof(double));
    hipMalloc(&d_Yn, nx * ny * sizeof(double));
    hipMalloc(&d_v, nx * ny * sizeof(double));
   
    hipMemcpy(d_u, u, nx * ny * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_v, v, nx * ny * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Yn, Yn, nx * ny * sizeof(double), hipMemcpyHostToDevice);

    dim3 blockDim(8,8);
    dim3 gridDim((nx + blockDim.x - 1) / blockDim.x, (ny + blockDim.y - 1) / blockDim.y);
    computeB<<<gridDim, blockDim>>>(d_b, d_Yn, d_u, d_v, dx, dy, nx, ny, dt);
    hipMemcpy(b, d_b, nx * ny * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < nx*ny; i++)
    {
        printf("hold %f\n", b_expeted[i]);
        printf("new %f\n", b[i]);

        //    assert(fabs(x_new[i] - x[i]) < 0.5);
    }
    hipFree(d_b);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_Yn);
}