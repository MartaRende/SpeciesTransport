#include "../solve/tools.h"
#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cassert>

void runTestRowOffset(int * row, int nx, int ny, const char* testName) {
    int *h_row_offsets = new int[ny+1];
    int *d_row_offsets;
    hipMalloc(&d_row_offsets, (ny+1)* sizeof(int));

    dim3 blockDim(16, 16);
    dim3 gridDim((nx + blockDim.x - 1) / blockDim.x, (ny + blockDim.y - 1) / blockDim.y);

    initializeRowOffsetsKernel<<<gridDim, blockDim>>>(d_row_offsets, nx, ny);
    hipDeviceSynchronize();

    hipMemcpy(h_row_offsets, d_row_offsets,  (ny+1) * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < ny+1; ++i) {
            //row[idx] = h_row_offsets[idx];
            printf("%d %d\n",row[i], h_row_offsets[i]);
            //assert(h_row_offsets[idx] == 5 * idx);
        
    }

    std::cout << testName << " passed successfully." << std::endl;

    delete[] h_row_offsets;
    hipFree(d_row_offsets);
}
