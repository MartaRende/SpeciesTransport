#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <hip/hip_runtime.h>
#include <cmath>
#include "../solve/tools.h"
#include "unitTest.h"
#include <cmath>  


void testJacobiSolver(int nx, int ny, int nnz,int * row, int * col , double * values, double * b, double* x, double *x_new, const char *testName) {

    // Allocate device memory
    int *d_row, *d_col;
    double *d_values, *d_b, *d_x, *d_x_new;
    hipMalloc(&d_row,( ny+1) * sizeof(int));
    hipMalloc(&d_col, nnz* sizeof(int));
    hipMalloc(&d_values,nnz * sizeof(double));
    hipMalloc(&d_b, ny * sizeof(double));
    hipMalloc(&d_x, ny * sizeof(double));
    hipMalloc(&d_x_new, ny * sizeof(double));

    // Copy data to device
    hipMemcpy(d_row, row, ( ny+1)* sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col, col,nnz* sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, values,nnz* sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, ny * sizeof(double), hipMemcpyHostToDevice);
   hipMemcpy(d_x, x_new, ny * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x_new, x_new, ny * sizeof(double), hipMemcpyHostToDevice);
    hipMemset(d_x, 0, ny * sizeof(double));

    // Launch kernel
   dim3 blockDim(16, 16);
    dim3 gridDim((nx + blockDim.x - 1) / blockDim.x, (ny + blockDim.y - 1) / blockDim.y);
    jacobiKernel<<<gridDim, blockDim>>>(d_row, d_col, d_values, d_b, d_x, d_x_new, nx, ny, nnz, 100, 1e-6);

    // Copy result back to host
    hipMemcpy(x_new, d_x, ny * sizeof(double), hipMemcpyDeviceToHost);
hipDeviceSynchronize();
    // Check result
    for(int i = 0; i<ny;i++){
   
 if (fabs(x_new[i] - x[i]) >= 0.5) {
            throw std::runtime_error("Assertion failed: Values are not within tolerance!");
        }
    }

    // Free device memory
    hipFree(d_row);
    hipFree(d_col);
    hipFree(d_values);
    hipFree(d_b);
    hipFree(d_x);
    hipFree(d_x_new);

    std::cout << testName << " passed successfully." << std::endl;

}
