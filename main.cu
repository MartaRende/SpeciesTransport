#include <iostream>
#include <sstream>
#include <string>
#include <sys/stat.h>
#include <mpi.h>
#include "solve/solve.h"
#include "solve/tools.h"

#include "write/write.h"
#include "initialization/init.h"
#include <hip/hip_runtime.h>
#include <chrono>
#include "./common_includes.c"
using namespace std;
using namespace std::chrono;

int main(int argc, char *argv[])
{

    // == MPI Initialization ==
    MPI_Status status;
    int world_size, world_rank;

    MPI_Init(&argc, &argv);
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);
    MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);
    auto start_total = high_resolution_clock::now();

    //  == Number of species to be calculated ==
    int nSpecies = 1;

    // == Output ==
    string outputName = "output/speciesTransport_";
    int count = 0; // to print file number

    // == Spatial parameters ==
    double D[nSpecies] = {0.005, 0.002, 0.010, 0.005, 0.015, 0.020}; // possible values from 0.001 to 0.025, each specie has its own diffusion coefficient
    int nx = 36;                                                     // in parallel 800
    int ny = 36;                                                     // in parallel 800
    double Lx = 1.0;
    double Ly = 1.0;
    double dx = Lx / (nx - 1); // in final version 0.0077
    double dy = Ly / (ny - 1); // in final version 0.0077

    // == Temporal ==
    double tFinal = 2.0;
    double dt = 0.0005;
    int nSteps = int(tFinal / dt);

    // == Variables to compute mean of file writing ==
    double meanFileWriting = 0;
    int totFileWrited = 40;

    // == Host variables ==

    int unidimensional_size = nx * ny;
    int unidimensional_size_of_bytes = unidimensional_size * sizeof(double);
    size_t nnz_estimate = nx * ny * 5;

    double **Y = (double **)malloc(nSpecies * sizeof(double *)); // Y is a 2d array because in y will be the divided results of the 6 species to be calculated
    for (int s = 0; s < nSpecies; s++)
    {
        Y[s] = (double *)malloc(unidimensional_size_of_bytes);
    }

    double *u = (double *)malloc(unidimensional_size_of_bytes);
    double *v = (double *)malloc(unidimensional_size_of_bytes);
    int *arrStart = new int[world_size];
    int *arrEnd = new int[world_size];
    int *splittedLengthes = new int[world_size];

    // == Calculate how to split the array for the mpi part ==
    if (world_rank == 0)
    {
        long int rest = unidimensional_size % world_size;
        long int nbrOfElements = unidimensional_size / world_size;
        for (int i = 0; i < world_size; i++)
        {
            if (i < rest)
            {
                arrStart[i] = i * (nbrOfElements + 1);
                arrEnd[i] = (i + 1) * (nbrOfElements + 1);
                splittedLengthes[i] = (nbrOfElements + 1);
            }
            else
            {
                arrStart[i] = rest * (nbrOfElements + 1) + (i - rest) * nbrOfElements;
                arrEnd[i] = rest * (nbrOfElements + 1) + (i - rest + 1) * nbrOfElements;
                splittedLengthes[i] = nbrOfElements;
            }
        }
    }

    MPI_Bcast(arrStart, world_size, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Bcast(arrEnd, world_size, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Bcast(splittedLengthes, world_size, MPI_INT, 0, MPI_COMM_WORLD);

    double **Y_splietted = new double *[nSpecies]; // Allocate array of pointers for species
    double *u_splitted = new double[splittedLengthes[world_rank]];
    double *v_splitted = new double[splittedLengthes[world_rank]];
    for (int s = 0; s < nSpecies; s++)
    {
        Y_splietted[s] = new double[splittedLengthes[world_rank]]; // allocate each species' array
    }

    // == CUDA part initialisation ==
    double *d_Y, *d_u, *d_v;
    double *d_Yn, *d_x, *d_values, *d_x_new, *d_b_flatten;
    int *d_column_indices, *d_row_offsets;

    if (world_rank == 0)
    {
        hipMalloc((void **)&d_Y, unidimensional_size_of_bytes); // for initialisation part d_yn could have been used
        hipMalloc((void **)&d_u, unidimensional_size_of_bytes); // velocity field u
        hipMalloc((void **)&d_v, unidimensional_size_of_bytes);// velocity field v 
        CHECK_ERROR(hipMalloc((void **)&d_Yn, nSpecies * unidimensional_size_of_bytes)); //
        CHECK_ERROR(hipMalloc((void **)&d_x, nSpecies * unidimensional_size_of_bytes));
        CHECK_ERROR(hipMalloc((void **)&d_x_new, nSpecies * nx * ny * sizeof(double)));
        CHECK_ERROR(hipMalloc((void **)&d_b_flatten, unidimensional_size_of_bytes));
        CHECK_ERROR(hipMalloc((void **)&d_values, nnz_estimate * sizeof(double)));
        CHECK_ERROR(hipMalloc((void **)&d_column_indices, nnz_estimate * sizeof(int)));
        CHECK_ERROR(hipMalloc((void **)&d_row_offsets, (nx * ny + 1) * sizeof(int)));
    }
    // == initialisation of the simulation ==
    if (world_rank == 0)
    {
        for (int s = 0; s < nSpecies; s++)
        {
            Initialization(Y[s], u, v, nx, ny, dx, dy, s, d_Y, d_u, d_v);
            dim3 blockDim(16, 16); // kernel size 2d
            dim3 gridDim((nx + blockDim.x - 1) / blockDim.x, (ny + blockDim.y - 1) / blockDim.y);
            computeBoundariesKernel<<<gridDim, blockDim>>>(d_Y, nx, ny);
            // copy into the host variables results obtained from initialisation
            hipMemcpy(Y[s], d_Y, unidimensional_size_of_bytes, hipMemcpyDeviceToHost); 
            hipMemcpy(u, d_u, unidimensional_size_of_bytes, hipMemcpyDeviceToHost);
            hipMemcpy(v, d_v, unidimensional_size_of_bytes, hipMemcpyDeviceToHost);
        }
    }
// == proc 0 send variables to be writed into files and the other processes receive the variables
    if (world_rank == 0)
    {

        for (int i = 1; i < world_size; i++)
        {
            for (int s = 0; s < nSpecies; s++)
            {
                MPI_Send(Y[s] + arrStart[i], splittedLengthes[i], MPI_DOUBLE, i, 0, MPI_COMM_WORLD);
            }
            MPI_Send(u + arrStart[i], splittedLengthes[i], MPI_DOUBLE, i, 0, MPI_COMM_WORLD);
            MPI_Send(v + arrStart[i], splittedLengthes[i], MPI_DOUBLE, i, 0, MPI_COMM_WORLD);
        }
        Y_splietted = Y;
        u_splitted = u;
        v_splitted = v;
    }
    else
    {
        for (int s = 0; s < nSpecies; s++)
        {
            MPI_Recv(Y_splietted[s], splittedLengthes[world_rank], MPI_DOUBLE, 0, 0, MPI_COMM_WORLD, &status);
        }
        MPI_Recv(u_splitted, splittedLengthes[world_rank], MPI_DOUBLE, 0, 0, MPI_COMM_WORLD, &status);
        MPI_Recv(v_splitted, splittedLengthes[world_rank], MPI_DOUBLE, 0, 0, MPI_COMM_WORLD, &status);
    }
// == conversions needed to write the different parts of the variables correctly into the files ==
    string WriteU = getString(u_splitted, splittedLengthes[world_rank], world_rank);
    string WriteV = getString(v_splitted, splittedLengthes[world_rank], world_rank);

    string *WriteY = new string[nSpecies];

    for (int s = 0; s < nSpecies; s++)
    {
        WriteY[s] = getString(Y_splietted[s], splittedLengthes[world_rank], world_rank);
    }
    // == write initialisation vtk file ==
    writeDataVTK(outputName, WriteY, WriteU, WriteV, nx, ny, dx, dy, count++, world_rank, world_size, nSpecies);
    auto end_init = high_resolution_clock::now();

    auto initDuration = chrono::duration_cast<chrono::microseconds>(end_init - start_total).count(); // Calculate init duration
    auto start_loop = high_resolution_clock::now();
    // == Part to solve species equation ==
    for (int step = 1; step <= nSteps; step++)
    {
        if (world_rank == 0)
        {
            for (int s = 0; s < nSpecies; s++)
            {
                // == copy the results of Y obtained from the initialization into d_Yn == 
                if (step == 1)
                {
                    CHECK_ERROR(hipMemcpy(&d_Yn[s * nx * ny], Y[s], unidimensional_size_of_bytes, hipMemcpyHostToDevice))
                }
                solveSpeciesEquation(Y[s], dx, dy, D[s], nx, ny, dt, d_u, d_v, &d_Yn[s * nx * ny], d_x, &d_x_new[s * nx * ny], d_b_flatten, d_values, d_column_indices, d_row_offsets, world_rank);
                if (step % 100 == 0)
                {
                    CHECK_ERROR(hipMemcpy(Y[s], &d_x[s * nx * ny], unidimensional_size_of_bytes, hipMemcpyDeviceToHost));
                }
            }
            for (int i = 1; i < world_size; i++)
            {
                for (int s = 0; s < nSpecies; s++)
                {
                    MPI_Send(Y[s] + arrStart[i], splittedLengthes[i], MPI_DOUBLE, i, 0, MPI_COMM_WORLD);
                }
            }
            Y_splietted = Y;
        }
        else
        {
            for (int s = 0; s < nSpecies; s++)
            {
                MPI_Recv(Y_splietted[s], splittedLengthes[world_rank], MPI_DOUBLE, 0, 0, MPI_COMM_WORLD, &status);
            }
        }

        if (step % 100 == 0)
        {
            auto start_write = high_resolution_clock::now();

            string *WriteY = new string[nSpecies];

            for (int s = 0; s < nSpecies; s++)
            {

                WriteY[s] = getString(Y_splietted[s], splittedLengthes[world_rank], world_rank);
            }

            writeDataVTK(outputName, WriteY, WriteU, WriteV, nx, ny, dx, dy, count++, world_rank, world_size, nSpecies);
            auto end_write = chrono::duration_cast<microseconds>(high_resolution_clock::now() - start_write).count();
            if (world_rank == 0)
            {
                printf("[MAIN] Write file %d took: %ld us\n", count, end_write);

                meanFileWriting += (double)end_write;
            }
        }
    }
    auto end_loop = high_resolution_clock::now();
    auto loopDuration = duration_cast<microseconds>(end_loop - start_loop).count(); // Calculate loop duration

    // Free memory using free()
    for (int s = 0; s < nSpecies; s++)
    {
        free(Y[s]); // Free the pointer to the array of rows for each species
    }
    free(Y); // Free the pointer to the array of species

    free(u); // Free the pointer to the array of u rows
    free(v); // Free the pointer to the array of v rows
    if (world_rank == 0)
    {
        // Free memory
        hipFree(d_Yn);
        hipFree(d_x);
        hipFree(d_b_flatten);

        hipFree(d_values);
        hipFree(d_column_indices);
        hipFree(d_row_offsets);
        hipFree(d_Y);
        hipFree(d_u);
        hipFree(d_v);
        auto end_total = high_resolution_clock::now();
        auto totalDuration = duration_cast<microseconds>(end_total - start_total).count(); // Calculate total duration
        meanFileWriting /= totFileWrited;

        printf("[MAIN] Initialization took: %ld us\n", initDuration);

        printf("[MAIN] Loop took : %ld us\n", loopDuration);

        printf("[MAIN] Mean of file Writing %f us \n", meanFileWriting);
        printf("[MAIN] Total time taken: %ld us\n", totalDuration);
    }

    MPI_Finalize();
    return 0;
}
