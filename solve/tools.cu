#include "hip/hip_runtime.h"
#include <iostream>
#include "tools.h"
__global__ void jacobiKernel(int *row, int *col, double *value, double *b, double *x, double *x_new, int nx, int ny, int nnz, int max_iterations, double tolerance)
{
    // 2D block and grid dimensions
    int j = blockIdx.x * blockDim.x + threadIdx.x; // Column index
    int i = blockIdx.y * blockDim.y + threadIdx.y; // Row index

    // Check if thread is within bounds
    if (i < ny && j < nx)
    {
        int idx = i * nx + j;  // 2D index flattened to 1D

        for (int iter = 0; iter < max_iterations; ++iter)
        {
            double sum = 0.0;
            double diag = 1.0;
            int row_start = row[j]; // Starting index for this row in the sparse matrix
            int row_end = row[j + 1]; // Ending index for this row

            // Calculate the sum and diagonal for Jacobi iteration
            for (int k = row_start; k < row_end; k++)
            {               
                   // printf("%f %d\n",value[k],idx);

                if (col[k] == idx) // Diagonal element
                {    

                                
                    diag = value[k];
                   if(value[k]==0.0){
               /*          printf("%d %d\n",row_start,row_end);
               
               
                printf("%d\n",value[k]); */

                    } 


                }
                else // Off-diagonal elements
                {
                    sum += value[k] * x_new[col[k]]; // Use x_new for the previous iteration

                }

            }

            // Calculate the new value for this element in the grid
            double new_value = (b[idx] - sum) / diag;

            // Update the new value for x_new
           
            // Check for convergence (based on the tolerance)
            if (fabs(new_value - x_new[idx]) < tolerance)
            {
                    x_new[idx] = new_value;

                break;
            }else{
    x_new[idx] = new_value;
            }
                     


            // Synchronize threads before next iteration (not strictly necessary in this case)
           // __syncthreads();
        }
    }
}


__global__ void fillMatrixAKernel(double *values, int *column_indices, int *row_offsets,
                                   const double dx, const double dy, const double D,
                                   const double dt, const int nx, const int ny)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x; // Column index
    int i = blockIdx.y * blockDim.y + threadIdx.y; // Row index

if (i == 0 || i == ny - 1 || j == 0 || j == nx - 1 )
    return;

    int idx = i * nx + j;  // Flattened index

    // Calculate the number of non-zero elements in the current row
    int num_non_zero_elements = 1;  // Diagonal element (always non-zero)
    if (j > 0) num_non_zero_elements++;  // Left neighbor
    if (j < nx - 1) num_non_zero_elements++; // Right neighbor
    if (i > 0) num_non_zero_elements++; // Top neighbor
    if (i < ny - 1) num_non_zero_elements++; // Bottom neighbor

    // Update row_offsets[idx] to track the start of this row in the sparse matrix
    int row_start = row_offsets[idx];
    
    // Debugging: Print the row start and num_non_zero_elements
   // printf("idx: %d, row_start: %d, num_non_zero_elements: %d\n", idx, row_start, num_non_zero_elements);

    // Store the non-zero elements in values and column_indices for the current row
    int count = 0; // Counter to keep track of the number of elements added to this row

    // Diagonal (current element)
    double diag_val = 1 + dt * D * (2 / (dx * dx) + 2 / (dy * dy));
    values[row_start + count] = diag_val;  // Store diagonal value
    column_indices[row_start + count] = idx; // Store column index for diagonal element
    count++; // Increment the count of non-zero elements

    // Debugging: Print the diagonal value and its position
    //printf("Diagonal value for idx %d: %f, at position %d\n", idx, diag_val, row_start + count - 1);

    // Left neighbor
    if (j > 0) {
        values[row_start + count] = -dt * D / (dx * dx);
        column_indices[row_start + count] = idx - 1; // Left neighbor
        count++;
    }

    // Right neighbor
    if (j < nx - 1) {
        values[row_start + count] = -dt * D / (dx * dx);
        column_indices[row_start + count] = idx + 1; // Right neighbor
        count++;
    }

    // Top neighbor
    if (i > 0) {
        values[row_start + count] = -dt * D / (dy * dy);
        column_indices[row_start + count] = idx - nx; // Top neighbor
        count++;
    }

    // Bottom neighbor
    if (i < ny - 1) {
        values[row_start + count] = -dt * D / (dy * dy);
        column_indices[row_start + count] = idx + nx; // Bottom neighbor
        count++;
    }



      //row_offsets[idx + 1] = row_offsets[idx] + num_non_zero_elements;
    
}



__global__ void computeB(double *b, double *Y_n, double *u, double *v,
                         const double dx, const double dy, const int nx, const int ny, const double dt)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i == 0 || i == ny - 1 || j == 0 || j == nx - 1)
        return;

    int idx = i * nx + j;
    int right = i * nx + (j + 1);
    int left = i * nx + (j - 1);
    int top = (i - 1) * nx + j;
    int down = (i + 1) * nx + j;

    b[idx] = Y_n[idx];

    if (u[idx] < 0.0)
        b[idx] -= dt * (u[idx] * (Y_n[down] - Y_n[idx]) / dx);
    else
        b[idx] -= dt * (u[idx] * (Y_n[idx] - Y_n[top]) / dx);

    if (v[idx] < 0.0)
        b[idx] -= dt * (v[idx] * (Y_n[right] - Y_n[idx]) / dy);
    else
        b[idx] -= dt * (v[idx] * (Y_n[idx] - Y_n[left]) / dy);
    //  printf("%f\n", Y_n[idx]);
}
__global__ void initializeRowOffsetsKernel(int *row_offsets, const int nx, const int ny)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Row index

    if (i >= ny+ 1)
        return; // Bounds check

    int offset = 0;
    for (int row = 0; row <= i; row++) {
        int num_non_zero_elements;
        if (row == 0 || row == ny-1) {
            num_non_zero_elements = 3 ;
        }
        else if (row == 1 || row == ny - 2) {
            num_non_zero_elements = 4;
        }
        else {
            num_non_zero_elements = 5;
        }
        
        if (row == i) {
            row_offsets[i] = offset;
        }
        offset += 5;
    }
}

