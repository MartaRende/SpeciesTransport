#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <unordered_map>
#include "../common_includes.c"
#include <chrono>
using namespace std;
using namespace chrono;
#include <vector>
#include <hip/hip_runtime.h>
#include "solve.h"
#include "tools.h"

__global__ void fillMatrixAKernel(double *values, int *column_indices, int *row_offsets,
                                  const double dx, const double dy, const double D,
                                  const double dt, const int nx, const int ny)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= nx || j >= ny || i == 0 || i == nx - 1 || j == 0 || j == ny - 1)
        return;

    int idx = i * ny + j;
    int count = 0;

    int row_start = row_offsets[idx];

    // Diagonal
    values[row_start + count] = 1 + dt * D * (2 / (dx * dx) + 2 / (dy * dy));
    column_indices[row_start + count++] = idx;

    // Left Neighbor
    values[row_start + count] = -dt * D / (dx * dx);
    column_indices[row_start + count++] = idx - ny;

    // Right Neighbor
    values[row_start + count] = -dt * D / (dx * dx);
    column_indices[row_start + count++] = idx + ny;

    // Top Neighbor
    values[row_start + count] = -dt * D / (dy * dy);
    column_indices[row_start + count++] = idx - 1;

    // Bottom Neighbor
    values[row_start + count] = -dt * D / (dy * dy);
    column_indices[row_start + count++] = idx + 1;
}

__global__ void computeB(double *b, double *Y_n, double *u, double *v,
                         const double dx, const double dy, const int nx, const int ny, const double dt)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i == 0 || i == nx - 1 || j == 0 || j == ny - 1)
        return;

    int idx = i * ny + j;
    int right = i * ny + (j + 1);
    int left = i * ny + (j - 1);
    int top = (i - 1) * ny + j;
    int down = (i + 1) * ny + j;

    b[idx] = Y_n[idx];

    if (u[idx] < 0.0)
        b[idx] -= dt * (u[idx] * (Y_n[down] - Y_n[idx]) / dx);
    else
        b[idx] -= dt * (u[idx] * (Y_n[idx] - Y_n[top]) / dx);

    if (v[idx] < 0.0)
        b[idx] -= dt * (v[idx] * (Y_n[right] - Y_n[idx]) / dy);
    else
        b[idx] -= dt * (v[idx] * (Y_n[idx] - Y_n[left]) / dy);
}
void solveSpeciesEquation(double *Y, double *u, double *v,
                          const double dx, const double dy, double D,
                          const int nx, const int ny, const double dt)
{
    auto start_total_solve = high_resolution_clock::now();

    int max_iter = 1000;
    double tol = 1e-3;
    size_t unidimensional_size_bytes = nx * ny * sizeof(double);
    size_t nnz_estimate = nx * ny * 5;

    // Allocate host memory
    double *Y_n = (double *)malloc(unidimensional_size_bytes);
    double *x = (double *)malloc(unidimensional_size_bytes);
    double *b_flatten = (double *)malloc(unidimensional_size_bytes);
    

    SparseMatrix A;
    A.row = (int *)malloc((nx * ny + 1) * sizeof(int));
    A.col = (int *)malloc(nnz_estimate * sizeof(int));
    A.value = (double *)malloc(nnz_estimate * sizeof(double));

    // Flatten input arrays
    for (int i = 0; i < nx*ny; i++)
    {
    
            Y_n[i ] = Y[i];
          
        
    }

    // Allocate device memory
    double *d_Yn, *d_x, *d_u, *d_v;
    double *d_values, *d_x_old, *d_x_new, *d_partial_diff, *d_diff, *d_b_flatten;
    int *d_column_indices, *d_row_offsets;
    CHECK_ERROR(hipMalloc((void **)&d_Yn, unidimensional_size_bytes));
    CHECK_ERROR(hipMalloc((void **)&d_x, unidimensional_size_bytes));
    CHECK_ERROR(hipMalloc((void **)&d_x_old, unidimensional_size_bytes));
    CHECK_ERROR(hipMalloc((void **)&d_x_new, unidimensional_size_bytes));
    CHECK_ERROR(hipMalloc((void **)&d_b_flatten, unidimensional_size_bytes));
    CHECK_ERROR(hipMalloc((void **)&d_u, unidimensional_size_bytes));
    CHECK_ERROR(hipMalloc((void **)&d_v, unidimensional_size_bytes));
    CHECK_ERROR(hipMalloc((void **)&d_values, nnz_estimate * sizeof(double)));
    CHECK_ERROR(hipMalloc((void **)&d_column_indices, nnz_estimate * sizeof(int)));
    CHECK_ERROR(hipMalloc((void **)&d_row_offsets, (nx * ny + 1) * sizeof(int)));
    CHECK_ERROR(hipMalloc((void **)&d_diff, nx * ny * sizeof(double)));

    // Copy input data to device
    CHECK_ERROR(hipMemcpy(d_Yn, Y_n, unidimensional_size_bytes, hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpy(d_u, u, unidimensional_size_bytes, hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpy(d_v, v, unidimensional_size_bytes, hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemset(d_diff, 0, nx*ny * sizeof(double)));
    hipMemset(d_x_new, 0, nx * ny * sizeof(double));


    dim3 blockDim(10, 10);
    dim3 gridDim((nx + blockDim.x - 1) / blockDim.x, (ny + blockDim.y - 1) / blockDim.y);
    int threadsPerBlock = 256; 
    int numBlocks = (nx * ny + threadsPerBlock - 1) / threadsPerBlock;

    double diff = 0.0;
    auto end_init_solve = duration_cast<microseconds>(high_resolution_clock::now() - start_total_solve).count();
    printf("[SOLVE] Initialization took: %ld us\n", end_init_solve);
    auto start_fillMatrix = high_resolution_clock::now();

    // Fill A
    fillMatrixAKernel<<<gridDim, blockDim>>>(d_values, d_column_indices, d_row_offsets, dx, dy, D, dt, nx, ny);
    auto end_fillMatrix = duration_cast<microseconds>(high_resolution_clock::now() - start_fillMatrix).count();
    printf("[SOLVE] Fill Matrix A took: %ld us\n", end_fillMatrix);
    // hipDeviceSynchronize();

    // Compute b
    auto start_fillb = high_resolution_clock::now();
    computeB<<<gridDim, blockDim>>>(d_b_flatten, d_Yn, d_u, d_v, dx, dy, nx, ny, dt);
  
    hipDeviceSynchronize();

    CHECK_ERROR(hipMemcpy(b_flatten, d_b_flatten, unidimensional_size_bytes, hipMemcpyDeviceToHost));

    auto end_fillb = duration_cast<microseconds>(high_resolution_clock::now() - start_fillb).count();
    auto start_computex = high_resolution_clock::now();

    // Jacobi Solver
    for (int iter = 0; iter < 1000; ++iter)
    {
        // Launch Jacobi kernel
        jacobiKernel<<<gridDim, blockDim>>>(d_row_offsets, d_column_indices, d_values, d_b_flatten, d_x, d_x_new, nx, ny, 5 * nx * ny);
        hipDeviceSynchronize();
        // Launch difference kernel
        diffKernel<<<gridDim, blockDim>>>(d_x, d_x_new, d_diff, nx ,ny);
        hipDeviceSynchronize();
        //hipMemcpy(d_x, d_x_new, nx * ny * sizeof(double), hipMemcpyDeviceToDevice);

        double *h_diff = new double[nx * ny];
        hipMemcpy(h_diff, d_diff, nx * ny * sizeof(double), hipMemcpyDeviceToHost);

        double total_diff = 0.0;
        for (int i = 0; i < nx * ny; ++i)
            total_diff += h_diff[i];
        delete[] h_diff;

        if (total_diff < tol)
            break;
    }

    auto end_computex = duration_cast<microseconds>(high_resolution_clock::now() - start_computex).count();
    printf("[SOLVE] Fill x took: %ld us\n", end_computex);
    // Free d_diff
    hipFree(d_diff);

    // Copy results back to host
    CHECK_ERROR(hipMemcpy(x, d_x_new, unidimensional_size_bytes, hipMemcpyDeviceToHost));
  
    // Update Y
for (int i = 1; i < nx - 1; i++) {
    for (int j = 1; j < ny - 1; j++) {
        int idx = i * ny + j; // Proper 1D index for (i, j)
        Y[idx] = x[idx];
    }
}


    computeBoundaries(Y, nx, ny);

    // Free device memory
    hipFree(d_Yn);
    hipFree(d_x);
    hipFree(d_b_flatten);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_values);
    hipFree(d_column_indices);
    hipFree(d_row_offsets);
    hipFree(d_partial_diff);

    // Free host memory
    free(Y_n);
    free(x);
    free(b_flatten);

    free(A.row);
    free(A.col);
    free(A.value);

    auto end_total_solve = duration_cast<microseconds>(high_resolution_clock::now() - start_total_solve).count();
    printf("[SOLVE] Total time taken: %ld us\n", end_total_solve);
}

void computeBoundaries(double *Y, const int nx, const int ny)
{
    for (int i = 0; i < nx; i++)
    {
        Y[i * ny + (ny - 1)] = 0.0; 
        Y[i * ny + 0] = 0.0; 
    }
    for (int j = 0; j < ny; j++)
    {
        Y[0*ny+j] = 0.0;
        Y[(nx - 1)*ny+j] = 0.0;
    }
}