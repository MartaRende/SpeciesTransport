#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <unordered_map>
#include "../common_includes.c"
#include <chrono>
using namespace std;
using namespace chrono;
#include <vector>
#include <hip/hip_runtime.h>
#include "solve.h"
#include "tools.h"

__global__ void fillMatrixAKernel(double *values, int *column_indices, int *row_offsets,
                                  const double dx, const double dy, const double D,
                                  const double dt, const int nx, const int ny)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= ny || j >= nx || i == 0 || i == ny - 1 || j == 0 || j == nx - 1)
        return;

    int idx = i * nx + j;
    int count = 0;

    int row_start = row_offsets[idx];

    // Diagonal
    values[row_start + count] = 1 + dt * D * (2 / (dx * dx) + 2 / (dy * dy));
    column_indices[row_start + count++] = idx;

    // Left Neighbor
    values[row_start + count] = -dt * D / (dx * dx);
    column_indices[row_start + count++] = idx - ny;

    // Right Neighbor
    values[row_start + count] = -dt * D / (dx * dx);
    column_indices[row_start + count++] = idx + ny;

    // Top Neighbor
    values[row_start + count] = -dt * D / (dy * dy);
    column_indices[row_start + count++] = idx - 1;

    // Bottom Neighbor
    values[row_start + count] = -dt * D / (dy * dy);
    column_indices[row_start + count++] = idx + 1;
}

__global__ void computeB(double *b, double *Y_n, double *u, double *v,
                         const double dx, const double dy, const int nx, const int ny, const double dt)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i == 0 || i == ny - 1 || j == 0 || j == nx - 1)
        return;

    int idx = i * nx + j;
    int right = i * nx + (j + 1);
    int left = i * nx + (j - 1);
    int top = (i - 1) * nx + j;
    int down = (i + 1) * nx + j;

    b[idx] = Y_n[idx];

    if (u[idx] < 0.0)
        b[idx] -= dt * (u[idx] * (Y_n[down] - Y_n[idx]) / dx);
    else
        b[idx] -= dt * (u[idx] * (Y_n[idx] - Y_n[top]) / dx);

    if (v[idx] < 0.0)
        b[idx] -= dt * (v[idx] * (Y_n[right] - Y_n[idx]) / dy);
    else
        b[idx] -= dt * (v[idx] * (Y_n[idx] - Y_n[left]) / dy);
}
void solveSpeciesEquation(double *Y, 
                          const double dx, const double dy, double D,
                          const int nx, const int ny, const double dt,double * d_u, double * d_v, double * d_Yn, double * d_x, double * d_x_new, double * d_b_flatten, double * d_values, int * d_column_indices, int * d_row_offsets)
{
    auto start_total_solve = high_resolution_clock::now();

    int max_iter = 1;
    double tol = 1e-3;
    size_t unidimensional_size_of_bytes = nx * ny * sizeof(double);
    size_t nnz_estimate = nx * ny * 5;

    // Allocate host memory
    double *Y_n = (double *)malloc(unidimensional_size_of_bytes);
    double *x = (double *)malloc(unidimensional_size_of_bytes);
    double *b_flatten = (double *)malloc(unidimensional_size_of_bytes);

    SparseMatrix A;
    A.row = (int *)malloc((nx * ny + 1) * sizeof(int));
    A.col = (int *)malloc(nnz_estimate * sizeof(int));
    A.value = (double *)malloc(nnz_estimate * sizeof(double));

    // Flatten input arrays
    for (int i = 0; i < nx * ny; i++)
    {

        Y_n[i] = Y[i];
    }



    // Copy input data to device
    CHECK_ERROR(hipMemcpy(d_Yn, Y_n, unidimensional_size_of_bytes, hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpy(d_x, d_Yn, unidimensional_size_of_bytes, hipMemcpyDeviceToDevice));
    hipMemset(d_x_new, 0, nx * ny * sizeof(double));

    dim3 blockDim(16, 16);
    dim3 gridDim((nx + blockDim.x - 1) / blockDim.x, (ny + blockDim.y - 1) / blockDim.y);


    auto end_init_solve = duration_cast<microseconds>(high_resolution_clock::now() - start_total_solve).count();
    printf("[SOLVE] Initialization took: %ld us\n", end_init_solve);
    auto start_fillMatrix = high_resolution_clock::now();

    // Fill A
    fillMatrixAKernel<<<gridDim, blockDim>>>(d_values, d_column_indices, d_row_offsets, dx, dy, D, dt, nx, ny);
    auto end_fillMatrix = duration_cast<microseconds>(high_resolution_clock::now() - start_fillMatrix).count();
    printf("[SOLVE] Fill Matrix A took: %ld us\n", end_fillMatrix);
    // hipDeviceSynchronize();

    // Compute b
    auto start_fillb = high_resolution_clock::now();
    computeB<<<gridDim, blockDim>>>(d_b_flatten, d_Yn, d_u, d_v, dx, dy, nx, ny, dt);

    hipDeviceSynchronize();
    CHECK_ERROR(hipMemcpy(b_flatten, d_b_flatten, unidimensional_size_of_bytes, hipMemcpyDeviceToHost));

    auto end_fillb = duration_cast<microseconds>(high_resolution_clock::now() - start_fillb).count();
    auto start_computex = high_resolution_clock::now();

    // Jacobi Solver
    for (int iter = 0; iter < 1; ++iter)
    {
        // Launch Jacobi kernel
        jacobiKernel<<<gridDim, blockDim>>>(d_row_offsets, d_column_indices, d_values, d_b_flatten, d_x, d_x_new, nx, ny, 5 * nx * ny);
        hipDeviceSynchronize();
        // Launch difference kernel
  //     dim3 blockDim(256);  // Or any other suitable block size
//dim3 gridDim((nx * ny + blockDim.x - 1) / blockDim.x);
//diffKernel<<<gridDim, blockDim>>>(d_x, d_x_new, d_diff, nx, ny);
    
     //   hipDeviceSynchronize();
    hipMemcpy(d_x, d_x_new, nx * ny * sizeof(double), hipMemcpyDeviceToDevice);

    
    }

    auto end_computex = duration_cast<microseconds>(high_resolution_clock::now() - start_computex).count();
    printf("[SOLVE] Fill x took: %ld us\n", end_computex);


    // Copy results back to host
    CHECK_ERROR(hipMemcpy(Y, d_x, unidimensional_size_of_bytes, hipMemcpyDeviceToHost));

    // Update Y
  
    computeBoundaries(Y, nx, ny);

   

    // Free host memory
    free(Y_n);
    free(x);
    free(b_flatten);

    free(A.row);
    free(A.col);
    free(A.value);

    auto end_total_solve = duration_cast<microseconds>(high_resolution_clock::now() - start_total_solve).count();
    printf("[SOLVE] Total time taken: %ld us\n", end_total_solve);
}

void computeBoundaries(double *Y, const int nx, const int ny)
{
    for (int i = 0; i < ny; i++)
    {
        Y[i * nx + (nx - 1)] = 0.0;
        Y[i * nx + 0] = 0.0;
    }
    for (int j = 0; j < nx; j++)
    {
        Y[0 * nx + j] = 0.0;
        Y[(ny - 1) * nx + j] = 0.0;
    }
}